#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!
//This code is created by Mohammad H Najafi in May 2023

#include "bmm.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// TILEX and TILEY are used to set the number of threads in a CUDA block
#define TILEX 16
#define TILEY 16


// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!

dim3 getDimGrid(const int m, const int n) {
        dim3 dimGrid(n/TILEX,n/TILEY);
        return dimGrid;
}
dim3 getDimBlock(const int m, const int n) {
        dim3 dimBlock(TILEX,TILEY);
        return dimBlock;
}
__global__ void kernelFunc(float* ad, float* bd, float* cd, const int m, const int n) {

        __shared__ float a_sub[TILEY][TILEX];
        __shared__ float b_sub[TILEY][TILEX];

        int row = by * TILEY + ty;
        int col = bx * TILEX + tx;

        float ans_sub = 0.0;
        for (int i = 0; i < n/TILEX; i++) {
                a_sub[ty][tx] = ad[row * n + i * TILEX + tx];
                b_sub[ty][tx] = bd[(i * TILEY + ty) * n + col];
                __syncthreads();

                for(int k = 0 ; k < TILEX ; k++){
                        ans_sub += a_sub[ty][k] * b_sub[k][tx];
                }

		__syncthreads();
        }
        cd[row*n + col] =ans_sub;
}
